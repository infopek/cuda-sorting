#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <iostream>
#include <stdio.h>

#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

#define CUDA_CALL( call )               \
{                                       \
	hipError_t result = call;              \
	if ( hipSuccess != result )            \
		std::cerr << "CUDA error " << result << " in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString( result ) << " (" << #call << ")" << std::endl;  \
}

constexpr int length = 14;
int arr[length]{ 6, 8, 1, 1, 4, 2, 9, 0, 2, 2, 5, 7, 8, 7 };

__device__ int dev_arr[length];

void printArr()
{
	std::copy(std::begin(arr), std::end(arr), std::ostream_iterator<int>(std::cout, " "));
	std::cout << '\n';
}

/// <summary>
/// Swaps 'a' and 'b' if 'b' is less than 'a'.
/// </summary>
__forceinline __device__ void compareAndSwap(int* a, int* b)
{
	if (*b < *a)
	{
		int temp = *a;
		*a = *b;
		*b = temp;
	}
}

__global__ void sortGPUSingleBlockCUDA(int phase)
{
	int idx = threadIdx.x;
	idx *= 2;
	if (phase == 0 && idx + 1 < length)
		compareAndSwap(&dev_arr[idx], &dev_arr[idx + 1]);
	if (phase == 1 && idx + 2 < length)
		compareAndSwap(&dev_arr[idx + 1], &dev_arr[idx + 2]);
}

void sortGPUSingleBlock()
{
	CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_arr), arr, sizeof(arr)));

	for (int i = 0; i < length; i++)
		sortGPUSingleBlockCUDA KERNEL_ARGS2(1, length / 2)(i % 2);

	CUDA_CALL(hipMemcpyFromSymbol(arr, HIP_SYMBOL(dev_arr), sizeof(arr)));
}

int main()
{
	std::cout << "Array before sorting: \n";
	printArr();

	sortGPUSingleBlock();

	std::cout << "Array after sorting: \n";
	printArr();
}
